//#include <omp.h>

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
__global__ void parallel1(int a, int** binaryTree, int** prefixsums)
{
    int b = threadIdx.x;
    int sum;
    sum = binaryTree[a+1][2*b] + binaryTree[a+1][2*b+1];
    binaryTree[a][b] = sum;
} 
__global__ void parallel2(int a, int** binaryTree, int** prefixsums) //pow(2, x) nodes
{
    int b = threadIdx.x;
    if(b == 0)
        prefixsums[a][b] = binaryTree[a][b];
    else if(b % 2 == 1)
        prefixsums[a][b] = prefixsums[a-1][(b-1)/2];
    else
        prefixsums[a][b] = prefixsums[a-1][b/2-1] + binaryTree[a][b];
}
int main()
{
  const int num = 8;

  int height;
  int binaryTreeSize = 0;
  if(num%2 == 0)
  {
      binaryTreeSize = 2*num - 1;
      height = (int)(log2((float)binaryTreeSize+2) + .5);
  }
  else
  {
      binaryTreeSize = 2*num;
      height = (int)(log2((float)binaryTreeSize+1) + .5);
  }

  printf("%d\n", height);

  int x, y;
  //Instantiate tree
  int** binaryTree;
  hipMalloc(&binaryTree, height * sizeof(int*));
  for(x = 0; x < height; x++)
  {
      hipMalloc(&binaryTree[x], num * sizeof(int));
  }
  int** prefixsums;
  hipMalloc(&prefixsums, height * sizeof(int*));
  for(x = 0; x < height; x++)
  {
      hipMalloc(&prefixsums[x], num * sizeof(int));
  }

  int nums[8] = {2, 4, 6, 3, 4, 1, 0, 2};

  for(x = 0; x < num; x++)
  {
    binaryTree[height-1][x] = nums[x];
  }
  for(x = height-2; x >= 0; x--)
  {
    parallel1<<<1, (int)pow(2,x)>>>(x, binaryTree, prefixsums); 
  }


  for(x = 0; x < height; x++)
  {
    //omp_set_num_threads(numnodes);
    //#pragma omp parallel
    parallel2<<<1, (int)pow(2, x)>>>(x, binaryTree, prefixsums);
  }

  for(x = 0; x < height; x++)
  {
    for(y = 0; y < pow(2, x); y++)
      printf("%d ", binaryTree[x][y]);
    printf("\n");
  }
  puts("");
  for(x = 0; x < height; x++)
  {
    for(y = 0; y < pow(2, x); y++)
      printf("%d ", prefixsums[x][y]);
    printf("\n");
  }
  puts("");
/**
  for(x = (int)pow(2, height) - 1; x < (int)pow(2, height) + num; x++)
  {
    printf("%d ", prefixsums[x]);
  }
  printf("\n");
**/

  return 0;
}
